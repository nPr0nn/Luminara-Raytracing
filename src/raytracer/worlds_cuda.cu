#include "hip/hip_runtime.h"

#include "objects/hittable.h"
#include "objects/sphere.h"
#include "objects/triangle.h"

#include "camera.h"
#include "materials.h"

typedef struct World {
  // Objects
  hittable **objects;  // d_list
  hittable **collider; // d_world

  // Camera
  Camera **camera;
 
  i32 pixel_samples;
  i32 ray_max_depth;
  
  // Sky Box
  vec3 sky_color1;
  vec3 sky_color2;
} World;

#define RND (hiprand_uniform(&local_rand_state))

//--------------------------------------------------------------------------------------------------
// World 1

__global__ void simple_world(hittable **d_list, hittable **d_world, Camera **d_camera, f32 aspect_ratio){
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    d_list[0] = new sphere(vec3(0, -100.5, -1), 100, new lambertian(vec3(0.1, 0.5, 0.1)));
 
    i32 i = 1;
    // spheres
    d_list[i++] = new sphere(vec3(0, 0, 0), 0.5, new lambertian(vec3(0.1, 0.2, 0.5)));  
    d_list[i++] = new sphere(vec3(1, 0, 0), 0.5, new metal(vec3(0.8, 0.6, 0.2), 0.0));
    d_list[i++] = new sphere(vec3(-1, 0, 0), 0.5, new dielectric(1.5));  

    // triangle 1
    vec3 v[3] = {vec3(0, 1, 0.5), vec3(1, 0, 0.5), vec3(-1, 0, 0.5)};
    vec3 n[3] = {cross(v[0], v[1]), cross(v[0], v[1]), cross(v[0], v[1])};
    d_list[i++]    = new triangle(v, n, new dielectric(1.5), true);

    // triangle 2
    vec3 v2[3] = {vec3(0.5, 1, 1), vec3(1.5, 0, 1), vec3(-0.5, 0, 1) };
    vec3 n2[3] = {cross(v[0], v[1]), cross(v[0], v[1]), cross(v[0], v[1])}; 
    d_list[i++]    = new triangle(v2, n2, new lambertian(vec3(0.1, 0.2, 0.5)), true);
 
    *d_world = new hittable_list(d_list, i);

    // Camera
    vec3 lookfrom     = vec3(-4, 3, -10); 
    vec3 lookat       = vec3(0, 0, 0);
    vec3 vup          = vec3(0, 1, 0);
    
    f32 vfov          = 20;
    f32 aperture      = 0.1;
    f32 focus_dist    = 10.0;
 
    *d_camera = new Camera(lookfrom, lookat, vup, vfov, aspect_ratio, aperture, focus_dist); 
  }
}

//--------------------------------------------------------------------------------------------------
// World 2
__global__ void book_cover_world(hittable **d_list, hittable **d_world, Camera **d_camera, f32 aspect_ratio, randState *rand_state) {

  if (threadIdx.x == 0 && blockIdx.x == 0) {
    randState local_rand_state = *rand_state;

    d_list[0] = new sphere(vec3(0, -1000.0, 0), 1000, new lambertian(vec3(0.5, 0.5, 0.5)));

    i32 i = 1;
    for (i32 a = -11; a < 11; a++) {
      for (i32 b = -11; b < 11; b++) {
        f32 choose_mat = RND;
        vec3 center(a + RND, 0.2, b + RND);
        if (choose_mat < 0.8f) {
          d_list[i++] =
              new sphere(center, 0.2, new lambertian(vec3(RND * RND, RND * RND, RND * RND)));
        } else if (choose_mat < 0.95f) {
          d_list[i++] = new sphere(center, 0.2,
              new metal(vec3(0.5f * (1.0f + RND), 0.5f * (1.0f + RND),
                             0.5f * (1.0f + RND)),
                        0.5f * RND));
        } else {
          d_list[i++] = new sphere(center, 0.2, new dielectric(1.5));
        }
      }
    }
    d_list[i++] = new sphere(vec3(0, 1, 0), 1.0, new dielectric(1.5));
    d_list[i++] = new sphere(vec3(-4, 1, 0), 1.0, new lambertian(vec3(0.4, 0.2, 0.1)));
    d_list[i++] = new sphere(vec3(4, 1, 0), 1.0, new metal(vec3(0.7, 0.6, 0.5), 0.0));
    *rand_state = local_rand_state;
    *d_world = new hittable_list(d_list, i);

    // Camera
    vec3 lookfrom     = vec3(13, 2, 3); 
    vec3 lookat       = vec3(0, 0, 0);
    vec3 vup          = vec3(0, 1, 0);
    
    f32 vfov          = 20;
    f32 aperture      = 0.1;
    f32 focus_dist    = 10.0;
 
    *d_camera = new Camera(lookfrom, lookat, vup, vfov, aspect_ratio, aperture, focus_dist);
    
  }
}

__global__ void free_world(hittable **d_list, hittable **d_world, Camera **d_camera) {
  for (i32 i = 0; i < WORLD_SPACE; i++) {
    delete ((sphere *)d_list[i])->mat_ptr;
    delete d_list[i];
  }
  delete *d_world;
  delete *d_camera;
}
