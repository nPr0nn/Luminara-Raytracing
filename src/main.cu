#include "hip/hip_runtime.h"

#include "../ext/glad/include/glad.h"
#include "../ext/glfw/include/GLFW/glfw3.h"

#include "raytracer/objects/hittable.h"
#include "utils/utils.h"
#include "window/windowContext.h"

#include <nvToolsExt.h>

#include "raytracer/geometry/ray.h"
#include "raytracer/geometry/vec3.h"
#include "raytracer/camera.h"

#include "raytracer/worlds_cuda.cu"

#include <hiprand/hiprand_kernel.h>
#include <time.h>

// limited version of checkCudaErrors
#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)
void inline check_cuda(hipError_t error, const char* func, const char* filename, i32 line) {
  if(error != hipSuccess) {
    fprintf(stderr, "Error: %s:%d: %s: %s\n", filename, line, hipGetErrorName(error), hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }  
}

// Substitute recursion for a for loop to avoid stack overflow
__device__ vec3 rayColor(const ray &r, World world, hiprandState *local_rand_state) {
  ray cur_ray = r;
  vec3 cur_attenuation = vec3(1.0, 1.0, 1.0);
  
  for (i32 i = 0; i < world.ray_max_depth; i++) {
    hit_record rec;
    if ((*(world.collider))->hit(cur_ray, 0.001f, INF, rec)) {
      ray scattered;
      vec3 attenuation;
      if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
        cur_attenuation = cur_attenuation * attenuation;
        cur_ray = scattered;
      } else {
        return vec3(0.0, 0.0, 0.0);
      }
    } else {
      vec3 unit_direction = normalize(cur_ray.direction());
      f32 t = 0.5f * (unit_direction.y() + 1.0f);
      vec3 c = (1.0f - t) * world.sky_color1 + t * world.sky_color2;
      return cur_attenuation * c;
    }
  }
  return vec3(0.0, 0.0, 0.0); // exceeded recursion
}

__global__ void rand_init(hiprandState *rand_state) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    hiprand_init(970, 0, 0, rand_state);
  }
}

__global__ void render_init(i32 width, int height, hiprandState *rand_state) {
  i32 i = threadIdx.x + blockIdx.x * blockDim.x;
  i32 j = threadIdx.y + blockIdx.y * blockDim.y;
  if ((i >= width) || (j >= height) || i < 0 || j < 0) return;
  
  i32 pixel_index = j * width + i;
  hiprand_init(970 + pixel_index, 0, 0, &rand_state[pixel_index]);
}

__global__ void rayTrace(vec3 *fb, i32 width, i32 height, World world, hiprandState *rand_state) {
  i32 i = threadIdx.x + blockIdx.x * blockDim.x;
  i32 j = threadIdx.y + blockIdx.y * blockDim.y;
  if ((i >= width) || (j >= height) || i < 0 || j < 0) return;
  
  i32 pixel_index = j * width + i;
  hiprandState local_rand_state = rand_state[pixel_index];
  
  vec3 col(0, 0, 0);
  for (i32 s = 0; s < world.pixel_samples; s++) {
    f32 u    = f32(i + hiprand_uniform(&local_rand_state)) / f32(width);
    f32 v    = f32(j + hiprand_uniform(&local_rand_state)) / f32(height);
    ray r    = (*world.camera)->get_ray(u, v, &local_rand_state);
    col      = col + rayColor(r, world, &local_rand_state);
  }
  
  rand_state[pixel_index] = local_rand_state;
  col      = col / f32(world.pixel_samples);
  col = vec3(sqrt(col.x()), sqrt(col.y()), sqrt(col.z()));
   
  fb[pixel_index] = col;
}

i32 main() {
  //------------------------------------
  // Init Window and Renderer
  //------------------------------------
  f32 aspect_ratio  = 16.0 / 9.0;
  i32 width         = 1200;
  i32 height        = (i32)(width / aspect_ratio);
  height = (height < 1) ? 1 : height;
  const char *title = "Luminara";

  i32 pixel_samples = 10;
  i32 ray_max_depth = 20; 
 
  WindowContext windowContext;
  windowContext.glfw_window = initWindowGLFW(width, height, title);
  initRenderer(&windowContext.renderer, width, height);
  const GLubyte* vendor = glGetString(GL_VENDOR);
  
  printf("OpenGL Vendor: %s\n", vendor);
  printf("Rendering a %dx%d image with %d samples per pixel.\n", width, height, pixel_samples);
  nvtxRangePush("rayTrace"); 
  
  //------------------------------------
  // World + Camera 
  //------------------------------------  
  // we need that a random state to be initialized for the world creation 
  hiprandState *d_rand_state_world;
  checkCudaErrors(hipMalloc((void **)&d_rand_state_world, 1 * sizeof(hiprandState)));
  rand_init<<<1, 1>>>(d_rand_state_world);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  // make our world of hittables objects and the main camera 
  World world;
  checkCudaErrors(hipMalloc((void **) &(world.objects), WORLD_SPACE * sizeof(hittable *)));  
  checkCudaErrors(hipMalloc((void **) &(world.collider), sizeof(hittable *))); 
  checkCudaErrors(hipMalloc((void **) &(world.camera), sizeof(Camera *))); 
  
  // simple_world<<<1, 1>>>(world.objects, world.collider, world.camera, aspect_ratio);  
  book_cover_world<<<1, 1>>>(world.objects, world.collider, world.camera, aspect_ratio, d_rand_state_world); 
  
  world.sky_color1     = vec3(1, 1, 1);
  world.sky_color2     = vec3(0.5, 0.7, 1.0);  
  
  world.pixel_samples  = pixel_samples;
  world.ray_max_depth  = ray_max_depth;
  
  //------------------------------------
  // Prepare Render Texture and run RayTracing
  //------------------------------------   
  // Allocate our frame buffer
  i32 num_pixels = width * height;
  vec3 *frame_buffer; 
  checkCudaErrors(hipMallocManaged((void **)&frame_buffer, num_pixels * sizeof(vec3))); 
  
  // Blocks and Threads
  i32 num_threads_x = 8;
  i32 num_threads_y = 8;  
  dim3 blocks(width / num_threads_x + 1, height / num_threads_y + 1);
  dim3 threads(num_threads_x, num_threads_y);
  
  // Allocate random state
  hiprandState *d_rand_state_trace;
  checkCudaErrors(hipMalloc((void **)&d_rand_state_trace, num_pixels * sizeof(hiprandState)));
  render_init<<<blocks, threads>>>(width, height, d_rand_state_trace);  

  // Raytrace
  printf("RayTracing...\n"); 
  clock_t start, stop;
  start = clock(); 
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  rayTrace<<<blocks, threads>>>(frame_buffer, width, height, world, d_rand_state_trace);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize()); 
  stop = clock();
  f64 timer_seconds = ((f64)(stop - start)) / CLOCKS_PER_SEC; 
  printf("Took %f s\n", timer_seconds);

  // Copy Frame Buffer into a Texture
  u8 *texture_data = (u8 *) malloc(width * height * 4);
  memset(texture_data, 0, width * height * 4); 
  for (i32 j = height - 1; j >= 0; j--) {
    for (i32 i = 0; i < width; i++) {
      i32 index = ((j)*width + i) * 4;
      i32 pixel_index = j * width + i;
      texture_data[index]     = (u8)(255.0f * frame_buffer[pixel_index].r());
      texture_data[index + 1] = (u8)(255.0f * frame_buffer[pixel_index].g());
      texture_data[index + 2] = (u8)(255.0f * frame_buffer[pixel_index].b());
      texture_data[index + 3] = 255;
    }
  }

  // Free CUDA Memory
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipFree(world.camera));
  checkCudaErrors(hipFree(world.collider));
  checkCudaErrors(hipFree(world.objects));
  checkCudaErrors(hipFree(d_rand_state_world));
  checkCudaErrors(hipFree(d_rand_state_trace));
  checkCudaErrors(hipFree(frame_buffer));
  hipDeviceReset(); 
  nvtxRangePop();

  //------------------------------------
  // Render to the window
  // TODO: Make it real time 
  //------------------------------------     
  while (!glfwWindowShouldClose(windowContext.glfw_window)) {
    glClear(GL_COLOR_BUFFER_BIT);
 
    // Render pixel data into a OpenGL texture
    glBindTexture(GL_TEXTURE_2D, windowContext.renderer.texture);
    glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, width, height, GL_RGBA, GL_UNSIGNED_BYTE, texture_data);

    glUseProgram(windowContext.renderer.shaderProgram);
    glBindTexture(GL_TEXTURE_2D, windowContext.renderer.texture);
    glBindVertexArray(windowContext.renderer.VAO);
    glDrawElements(GL_TRIANGLES, 6, GL_UNSIGNED_INT, 0);

    glfwSwapBuffers(windowContext.glfw_window);
    glfwPollEvents();
  }

  // Free Texture and Save Image to memory
  free(texture_data);
  saveRenderTexture(windowContext.renderer.texture, width, height, "raytraced_image.png");

  // Terminate Window Context data structures
  terminateRenderer(&windowContext.renderer);
  terminateWindowGLFW(windowContext.glfw_window);
  
}
